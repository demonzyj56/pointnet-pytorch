#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This is valid for num_points <= blockDim.x (=1024).
template <typename scalar_t>
__global__ void fps_cuda_kernel(int batch_size, int num_points, int num_centroids, const scalar_t *pcs,
        int64_t *centroid_idx) {
    // shared memory
    extern __shared__ float sdata[];
    float *max_dists = sdata;
    float *dists = &max_dists[blockDim.x];
    int64_t *max_idx = (int64_t *)&dists[blockDim.x];
    bool *sampled = (bool *)&max_idx[blockDim.x];

    int64_t batch = blockIdx.x;
    int64_t tid = threadIdx.x;

    // loop over all centroids
    centroid_idx[batch*num_centroids] = 0;
    dists[tid] = 1e38;
    sampled[tid] = (tid == 0) ? true : false;
    __syncthreads();
    for (int64_t i = 1; i < num_centroids; ++i) {
        int64_t prev_idx = centroid_idx[batch*num_centroids+i-1];
        if (tid < num_points) {
            scalar_t x1 = pcs[(batch*3+0)*num_points+prev_idx];
            scalar_t x2 = pcs[(batch*3+1)*num_points+prev_idx];
            scalar_t x3 = pcs[(batch*3+2)*num_points+prev_idx];
            scalar_t y1 = pcs[(batch*3+0)*num_points+tid];
            scalar_t y2 = pcs[(batch*3+1)*num_points+tid];
            scalar_t y3 = pcs[(batch*3+2)*num_points+tid];
            scalar_t d = static_cast<float>((x1-y1)*(x1-y1) + (x2-y2)*(x2-y2) + (x3-y3)*(x3-y3));
            if (d < dists[tid]) {
                dists[tid] = d;
            }
        } else {
            dists[tid] = -1.;
        }
        __syncthreads();

        max_dists[tid] = dists[tid];
        max_idx[tid] = tid;
        __syncthreads();

        // do reduction!
        for (int64_t s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s && tid < num_points) {
                // four cases depending whether each point is sampled or not.
                if (!sampled[max_idx[tid+s]]) {
                    if (sampled[max_idx[tid]] || (max_dists[tid] < max_dists[tid+s])) {
                        max_dists[tid] = max_dists[tid+s];
                        max_idx[tid] = max_idx[tid+s];
                    }
                }
            }
            __syncthreads();
        }

        // write result to global memory
        if (tid == 0) {
            centroid_idx[batch*num_centroids+i] = max_idx[0];
        }

        // update sampled record
        if (tid == max_idx[0]) {
            sampled[max_idx[0]] = true;
        }
        __syncthreads();

    }
}

template <typename scalar_t>
__global__ void fps_cuda_kernel_strided(int batch_size, int num_points, int num_centroids, const scalar_t *pcs,
        int64_t *centroid_idx, float *dist_buf, int *idx_buf) {
    extern __shared__ float sdata[];
    float *dists = sdata;  // distance record, size: num_points
    float *max_dists = &dists[num_points];  // current max distance buffer, size: blockDim.x
    int *max_idx = (int *)&max_dists[blockDim.x]; // current max idx buffer, size: blockDim.x
    bool *sampled = (bool *)&max_idx[blockDim.x]; // sampled record, size: num_points

    int batch = blockIdx.x;
    int tid = threadIdx.x;
    int stride = (num_points + blockDim.x - 1) / blockDim.x;
    dist_buf += batch;
    idx_buf += batch;

    // init
    if (tid == 0) {
        centroid_idx[batch*num_centroids] = 0;
    }
    for (int gid = tid; gid < num_points; gid += blockDim.x) {
        dists[gid] = 1e38;
        sampled[gid] = (gid == 0) ? true : false;
    }
    __syncthreads();
    // outer iteration loop
    for (int i = 1; i < num_centroids; ++i) {
        int prev_idx = centroid_idx[batch*num_centroids+i-1];
        scalar_t x1 = pcs[(batch*3+0)*num_points+prev_idx];
        scalar_t x2 = pcs[(batch*3+1)*num_points+prev_idx];
        scalar_t x3 = pcs[(batch*3+2)*num_points+prev_idx];
        // inner strided loop
        for (int gid = tid; gid < num_points; gid += blockDim.x) {
            scalar_t y1 = pcs[(batch*3+0)*num_points+gid];
            scalar_t y2 = pcs[(batch*3+1)*num_points+gid];
            scalar_t y3 = pcs[(batch*3+2)*num_points+gid];
            scalar_t d = static_cast<float>((x1-y1)*(x1-y1) + (x2-y2)*(x2-y2) + (x3-y3)*(x3-y3));
            if (d < dists[gid]) {
                dists[gid] = d;
            }
        }
        __syncthreads();
        
        // reset buffer
        if (tid == 0) {
            dist_buf[0] = -1.;
            idx_buf[0] = -1;
        }
        __syncthreads();
        // reduce over [gid, min(gid+blockDim.x, num_points))
        // Explicit specify current stride
        for (int j = 0; j < stride; ++j) {
            int gid = j * blockDim.x + tid;
            if (gid < num_points) {
                max_dists[tid] = dists[gid];
                max_idx[tid] = gid;
            }
            __syncthreads();

            // do reduction!
            for (int s = blockDim.x / 2; s > 0; s >>= 1) {
                if (tid < s && gid + s < num_points) {
                    if (!sampled[max_idx[tid+s]]) {
                        if (sampled[max_idx[tid]] || max_dists[tid] < max_dists[tid+s]) {
                            max_dists[tid] = max_dists[tid+s];
                            max_idx[tid] = max_idx[tid+s];
                        }
                    }
                }
                __syncthreads();
            }

            // compare to global buffer
            if (tid == 0 && !sampled[max_idx[0]]) {
                if (max_dists[0] > dist_buf[0]) {
                    dist_buf[0] = max_dists[0];
                    idx_buf[0] = max_idx[0];
                }
            }
            __syncthreads();
        }
        // write result to global memory
        if (tid == 0) {
            centroid_idx[batch*num_centroids+i] = idx_buf[0];
            sampled[idx_buf[0]] = true;
        }
        __syncthreads();
    }
}

void fps_cuda_strided(at::Tensor pcs, at::Tensor out) {
    int batch_size = pcs.size(0);
    int num_centroids = out.size(1);
    int num_points = pcs.size(2);
    int threads = 1024;
    int smem_size = (sizeof(float) + sizeof(int)) * threads + (sizeof(float) + sizeof(bool)) * num_points;
    float *d_buf = NULL;
    int *d_idx = NULL;
    hipMalloc((void **)&d_buf, batch_size*sizeof(float));
    hipMalloc((void **)&d_idx, batch_size*sizeof(int));
    AT_DISPATCH_FLOATING_TYPES(pcs.type(), "fps2_kernel", ([&] {
        fps_cuda_kernel_strided<<<batch_size, threads, smem_size>>>(batch_size, num_points, num_centroids,
                pcs.data<scalar_t>(), out.data<int64_t>(), d_buf, d_idx);
    }));
    hipFree(d_buf);
    hipFree(d_idx);
}

void fps_cuda(at::Tensor pcs, at::Tensor out) {
    int num_points = pcs.size(2);
    int threads = 1024;
    if (num_points > threads) {
        fps_cuda_strided(pcs, out);
    } else {
        int batch_size = pcs.size(0);
        int num_centroids = out.size(1);
        int smem_size = (2 * sizeof(float) + sizeof(int64_t) + sizeof(bool)) * threads;
        AT_DISPATCH_FLOATING_TYPES(pcs.type(), "fps1_kernel", ([&] {
            fps_cuda_kernel<<<batch_size, threads, smem_size>>>(batch_size, num_points, num_centroids,
                    pcs.data<scalar_t>(), out.data<int64_t>());
        }));
    }
}
